#include "hip/hip_runtime.h"
#include "../include/ssp_cuda.cuh"

__global__ void hellofunc()
{
    printf("Hello World\n");
}

void hello()
{
    hellofunc<<<1,1>>>();
    hipDeviceSynchronize();
}

void printCudaDevice(){
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("****** Using device %d ***********\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               devProps.name, (int)devProps.totalGlobalMem,
               (int)devProps.major, (int)devProps.minor,
               (int)devProps.clockRate);
        printf("Number of multiprocessors on device : %d\n", devProps.multiProcessorCount);
        printf("Maximum size of each dimension of a grid : %ld\n", devProps.maxGridSize);
        printf("Maximum size of each dimension of a block : %ld\n", devProps.maxThreadsDim);
        printf("Maximum number of threads per block : %d\n", devProps.maxThreadsPerBlock);
        //printf("Maximum number of resident blocks per multiprocessor : %d\n", devProps.maxBlocksPerMultiProcessor );
        printf("Maximum resident threads per multiprocessor : %d\n", devProps.maxThreadsPerMultiProcessor);
        printf("Shared memory available per block in bytes : %zu \n", devProps.sharedMemPerBlock );
        printf("Shared memory available per multiprocessor in bytes : %zu \n", devProps.sharedMemPerMultiprocessor );
        printf("Warp size in threads : %d \n", devProps.warpSize );
        printf("****** End of device stats ***********\n");
    }
}

__device__ __forceinline__ float atomicMinFloat (float * addr, float value) {
        float old;
        old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
             __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));

        return old;
}

__global__ void init_distances_cuda(const int n, float D[], const int source)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (index < n) 
    {
        D[index] = __FLT_MAX__;
        if(index == source-1)
            D[index] = 0.0;
    }
}

__global__ void updateIndexOfEdges_cuda(int num_nodes, int num_edges, int V[], int E[]) 
{
    int l,r;
    unsigned int index = threadIdx.x + (blockDim.x * blockIdx.x);

    // This does binary search on the V array to find the index of each node in the Edge array (E) and replace the same with index
    // Based on the iterative binary search from : https://www.geeksforgeeks.org/binary-search/
    if (index < num_edges) {
        l=0; r=num_nodes-1;
        while (l <= r) {
            int m = l + (r - l) / 2;
            // Check if x is present at mid
            if (V[m] == E[index]) {
                E[index] = m;
                break;
            }
            // If x greater, ignore left half
            if (V[m] < E[index]) {
                l = m + 1;
            } else {        // If x is smaller, ignore right half
                r = m - 1;
            }
        }
    }
}

__global__ void relax(int num_nodes, float MAX_VAL, int V[], int I[], int E[], float W[], float D[], float Di[]) 
{
    unsigned int index = threadIdx.x + (blockDim.x * blockIdx.x);

    if (index < num_nodes) 
    {
        for (int j = I[index]; j < I[index + 1]; j++) 
        {
            // int u = V[index];
            // int v = V[E[j]];
            float w = W[j];
            float du = D[index];
            float dv = D[E[j]];
            float newDist = du + w;
            if (du == MAX_VAL)
                newDist = MAX_VAL;
            // printf("Index = %d, w=%f, du =%.1e, dv=%.1e,  -- du + w = %.1e\n", index, w, du , dv, du + w);

            if (newDist < dv)
                atomicMinFloat(&Di[E[j]], newDist);
        }
    }
}


__global__ void update_distance(int num_nodes, float D[], float Di[]) 
{
    unsigned int index = threadIdx.x + (blockDim.x * blockIdx.x);
    if (index < num_nodes) 
    {
        if (D[index] > Di[index])
            D[index] = Di[index];
        Di[index] = D[index];

    }
}

void bellman_parallel(int source, int num_nodes, int num_edges, int V[], int I[], int E[], float W[], const char* filename)
{
    printf("\n Running BellmanFord Parallel for graph with %d nodes and %d edges and source node %d \n", num_nodes, num_edges, source);
    float *D = (float *) malloc(num_nodes * sizeof(float));
    int *device_V, *device_I, *device_E;
    float *device_W, *device_D, *device_Di;

    hipMalloc(&device_V, num_nodes*sizeof(int));
    hipMalloc(&device_I, (num_nodes+1)*sizeof(int));
    hipMalloc(&device_E, num_edges*sizeof(int));
    hipMalloc(&device_W, num_edges*sizeof(float));

    hipMalloc(&device_D, num_nodes*sizeof(float));
    hipMalloc(&device_Di, num_nodes*sizeof(float));

    hipMemcpy(device_V, V, num_nodes*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_I, I, (num_nodes+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_E, E, num_edges*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_W, W, num_edges*sizeof(float), hipMemcpyHostToDevice);

    int NUM_THREADS = 1024;
    int NUM_BLOCKS = ceil(num_edges/ (float) NUM_THREADS);
    init_distances_cuda<<<ceil(num_nodes/ (float) NUM_THREADS), NUM_THREADS>>>(num_nodes, device_D, source);
    init_distances_cuda<<<ceil(num_nodes/ (float) NUM_THREADS), NUM_THREADS>>>(num_nodes, device_Di, source);
    updateIndexOfEdges_cuda<<<ceil(num_edges/ (float) NUM_THREADS), NUM_THREADS>>>(num_nodes, num_edges, device_V, device_E);
    int *newE = (int *) malloc(num_edges * sizeof(int));

    for (int round = 1; round < num_nodes; round++)
    {
        relax<<<ceil(num_nodes/ (float) NUM_THREADS), NUM_THREADS>>>(num_nodes, __FLT_MAX__, device_V, device_I, device_E, device_W, device_D, device_Di);
        update_distance<<<ceil(num_nodes/ (float) NUM_THREADS), NUM_THREADS>>>(num_nodes, device_D, device_Di);
    }
    hipDeviceSynchronize();
    hipMemcpy(D, device_D, num_nodes*sizeof(float), hipMemcpyDeviceToHost);
    save_to_file(num_nodes, V, D, filename);

    hipFree(device_V);
    hipFree(device_I);
    hipFree(device_E);
    hipFree(device_W);
    hipFree(device_D);
    hipFree(device_Di);
}