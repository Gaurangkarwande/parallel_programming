#include "hip/hip_runtime.h"
#include "../include/ssp_cuda.cuh"

__global__ void hellofunc()
{
    printf("Hello World\n");
}

void hello()
{
    hellofunc<<<1,1>>>();
    hipDeviceSynchronize();
}

void printCudaDevice(){
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("****** Using device %d ***********\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               devProps.name, (int)devProps.totalGlobalMem,
               (int)devProps.major, (int)devProps.minor,
               (int)devProps.clockRate);
        printf("Number of multiprocessors on device : %d\n", devProps.multiProcessorCount);
        printf("Maximum size of each dimension of a grid : %ld\n", devProps.maxGridSize);
        printf("Maximum size of each dimension of a block : %ld\n", devProps.maxThreadsDim);
        printf("Maximum number of threads per block : %d\n", devProps.maxThreadsPerBlock);
        //printf("Maximum number of resident blocks per multiprocessor : %d\n", devProps.maxBlocksPerMultiProcessor );
        printf("Maximum resident threads per multiprocessor : %d\n", devProps.maxThreadsPerMultiProcessor);
        printf("Shared memory available per block in bytes : %zu \n", devProps.sharedMemPerBlock );
        printf("Shared memory available per multiprocessor in bytes : %zu \n", devProps.sharedMemPerMultiprocessor );
        printf("Warp size in threads : %d \n", devProps.warpSize );
        printf("****** End of device stats ***********\n");
    }
}

// int main()
// {
//     printCudaDevice();
//     return 0;
// }